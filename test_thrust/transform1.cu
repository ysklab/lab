#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const {
            return a * x + y;
        }
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    // Y <- A * X + Y
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void saxpy_slow(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::device_vector<float> temp(X.size());

    // temp <- A
    thrust::fill(temp.begin(), temp.end(), A);

    // temp <- A * X
    thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(), thrust::multiplies<float>());

    // Y <- A * X + Y
    thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(), thrust::plus<float>());
}

void main() {

  int sum = thrust::reduce(D.begin(), D.end(), (int) 0, thrust::plus<int>());
}